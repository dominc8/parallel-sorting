#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <string.h>

extern "C"
{
#include "utils.h"
}

FILE *log_file;

#ifndef NDEBUG

    #define LOG(format, ...) fprintf(log_file, format "\n", ##__VA_ARGS__)
    #define LOG_ARRAY(arr_ptr, size) print_array(log_file, (arr_ptr), (size))

#else

    #define LOG(format, ...) do {} while (0)
    #define LOG_ARRAY(arr_ptr, size) do {} while (0)

#endif /*NDEBUG*/

#define N_BLOCKS    512
#define N_THREADS   512

__global__ void gpu_sort(int32_t *arr, int32_t size);
static int32_t compare_arr(int32_t *arr1, int32_t *arr2, int32_t size);

int main(int argc, char **argv)
{
    const int32_t arr_size = (1<<4);
    int32_t diff_cnt;
    int32_t *arr;
    int32_t *arr_ref;

#ifdef LOG_TO_FILE
    log_file = fopen("logs", "w");
#else
    log_file = stdout;
#endif

    alloc_fill_array(&arr, arr_size, 2*arr_size);
    arr_ref = (int32_t *)malloc(arr_size * sizeof(*arr));
    memcpy(&arr_ref[0], &arr[0], arr_size * sizeof(*arr));
    LOG_ARRAY(&arr[0], arr_size);


    /* REF START */

    start_time();

    qsort(&arr_ref[0], arr_size, sizeof(*arr_ref), &cmpfunc_int32);

    stop_time();

    printf("[Ref qsort] Elapsed time: %lf ms\n", elapsed_time_ms());
    LOG("[Ref qsort] Sorted array");
    LOG_ARRAY(&arr_ref[0], arr_size);

    /* REF STOP */


    /* GPU START */
    start_time();

    gpu_sort<<<N_BLOCKS, N_THREADS>>>(&arr[0], arr_size);

    stop_time();

    printf("[CUDA] Elapsed time: %lf ms\n", elapsed_time_ms());
    LOG("[CUDA] Sorted array:");
    LOG_ARRAY(&arr[0], arr_size);

    /* GPU STOP */

    
    diff_cnt = compare_arr(&arr[0], &arr_ref[0], arr_size);

    if (diff_cnt != 0)
    {
        printf("Results differ on %d elements\n", diff_cnt);
    }
    else
    {
        printf("Arrays are the same!\n");
    }

    free(arr);
    free(arr_ref);


#ifdef LOG_TO_FILE
    fclose(log_file);
#endif

    return 0;
}

__global__ void gpu_sort(int32_t *arr, int32_t size)
{
}

static int32_t compare_arr(int32_t *arr1, int32_t *arr2, int32_t size)
{
    int32_t cnt = 0;
    int32_t i;
    for (i = 0; i < size; ++i)
    {
        if (arr1[i] != arr2[i])
            ++cnt;
    }
    return cnt;
}

