#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <string.h>

extern "C"
{
#include "utils.h"
}

FILE *log_file;

#ifndef NDEBUG

    #define LOG(format, ...) fprintf(log_file, format "\n", ##__VA_ARGS__)
    #define LOG_ARRAY(arr_ptr, size) print_array(log_file, (arr_ptr), (size))

#else

    #define LOG(format, ...) do {} while (0)
    #define LOG_ARRAY(arr_ptr, size) do {} while (0)

#endif /*NDEBUG*/

#define N_BLOCKS    (1<<15)
#define N_THREADS   (1<<9)
#define ARR_SIZE    ((N_BLOCKS)*(N_THREADS))

__global__ void gpu_sort(int32_t *arr, int32_t stage_idx, int32_t substage_idx);

void sort(int32_t *arr, int32_t size);
static int32_t compare_arr(int32_t *arr1, int32_t *arr2, int32_t size);

int main(int argc, char **argv)
{
    int32_t diff_cnt;
    int32_t *arr;
    int32_t *cuda_arr;
    int32_t *arr_ref;
    hipEvent_t start, stop, start_mem;
    float time, time_mem;

#ifdef LOG_TO_FILE
    log_file = fopen("logs", "w");
#else
    log_file = stdout;
#endif

    alloc_fill_array(&arr, ARR_SIZE, 2*ARR_SIZE);
    arr_ref = (int32_t *)malloc(ARR_SIZE * sizeof(*arr));
    memcpy(&arr_ref[0], &arr[0], ARR_SIZE * sizeof(*arr));
    LOG_ARRAY(&arr[0], ARR_SIZE);


    /* REF START */

    start_time();

    qsort(&arr_ref[0], ARR_SIZE, sizeof(*arr_ref), &cmpfunc_int32);

    stop_time();

    printf("[Ref qsort] Elapsed time: %lf ms\n", elapsed_time_ms());
    LOG("[Ref qsort] Sorted array");
    LOG_ARRAY(&arr_ref[0], ARR_SIZE);

    /* REF STOP */


    /* GPU START */
    hipEventCreate(&start_mem, 0);
    hipEventCreate(&start, 0);
    hipEventCreate(&stop, 0);

    hipEventRecord(start_mem, 0);

    hipMalloc((void **)&cuda_arr, sizeof(*cuda_arr) * ARR_SIZE);
    hipMemcpy(cuda_arr, arr, ARR_SIZE * sizeof(*arr), hipMemcpyHostToDevice);

    hipEventRecord(start, 0);

    sort(&cuda_arr[0], ARR_SIZE);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    hipEventElapsedTime(&time_mem, start_mem, stop);

    printf("[CUDA] Elapsed time: %f (%f) ms\n", time, time_mem);
    hipMemcpy(arr, cuda_arr, ARR_SIZE * sizeof(*arr), hipMemcpyDeviceToHost);
    LOG("[CUDA] Sorted array:");
    LOG_ARRAY(&arr[0], ARR_SIZE);

    /* GPU STOP */

    
    diff_cnt = compare_arr(&arr[0], &arr_ref[0], ARR_SIZE);

    if (diff_cnt != 0)
    {
        printf("Results differ on %d elements\n", diff_cnt);
    }
    else
    {
        printf("Arrays are the same!\n");
    }

    hipFree(cuda_arr);
    free(arr);
    free(arr_ref);


#ifdef LOG_TO_FILE
    fclose(log_file);
#endif

    return 0;
}

void sort(int32_t *arr, int32_t size)
{
    for (int32_t i = 2; i <= size; i <<= 1)
    {
        for (int32_t j = i>>1; j > 0; j >>= 1)
        {
            gpu_sort<<<N_BLOCKS, N_THREADS>>>(&arr[0], i, j);
            LOG("stage=%d, substage=%d", i, j);
        }
    }
}

__global__ void gpu_sort(int32_t *arr, int32_t stage_idx, int32_t substage_idx)
{
    int32_t idx = threadIdx.x + blockIdx.x * blockDim.x;
    int32_t idx_s = idx ^ substage_idx;

    if (idx_s > idx)
    {
        if (idx & stage_idx)
        {
            if (arr[idx] < arr[idx_s])
            {
                int32_t temp = arr[idx];
                arr[idx] = arr[idx_s];
                arr[idx_s] = temp;
            }
        }
        else
        {
            if (arr[idx] > arr[idx_s])
            {
                int32_t temp = arr[idx];
                arr[idx] = arr[idx_s];
                arr[idx_s] = temp;
            }
        }
    }
}

static int32_t compare_arr(int32_t *arr1, int32_t *arr2, int32_t size)
{
    int32_t cnt = 0;
    int32_t i;
    for (i = 0; i < size; ++i)
    {
        if (arr1[i] != arr2[i])
            ++cnt;
    }
    return cnt;
}

